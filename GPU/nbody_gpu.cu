#include <math.h>
#include <stdio.h>
#include <stdlib.h>
//#include "timer.h"
#include <hip/hip_runtime.h>
#include <assert.h>


#define MAX_BLOCKS_PER_DIM 65535
#define MIN(a,b) (((a) < (b))?(a):(b))
#define SOFTENING 1e-9f

typedef struct {
  float x, y, z;        /* particle positions */
  float vx, vy, vz;     /* particle momenta */
} Particle;


/* randomly initialize particle positions and momenta */
__host__ void ran_init(float *data, int n) {
  for (int i = 0; i < n; i++) {
    data[i] = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
  }
}


/* calculate all interparticle forces and update instantaneous velocities */
__host__ void calc_force(Particle *p, float dt, int n, int threads) {
  for (int i = 0; i < n; i++) {
    float Fx = 0.0f; float Fy = 0.0f; float Fz = 0.0f;

    for (int j = 0; j < n; j++) {
      /* calculate net particle force on i'th particle */
      if (j != i) {
        float dx = p[j].x - p[i].x;
        float dy = p[j].y - p[i].y;
        float dz = p[j].z - p[i].z;
        float distSqr = dx*dx + dy*dy + dz*dz + SOFTENING;
        float invDist = 1.0f / sqrtf(distSqr);
        float invDist3 = invDist * invDist * invDist;

        Fx += dx * invDist3; Fy += dy * invDist3; Fz += dz * invDist3;
      }
    }
    /* update instantaneous velocity based on force and timestep */
    p[i].vx += dt*Fx; p[i].vy += dt*Fy; p[i].vz += dt*Fz;
  }
}
__global__ void calc_force(Particle* p, int nParticles, float dt) {
    int idx = threadIdx.x + blockDim.x*blockIdx.x;
    if ( idx >= nParticles ) return;
    float Fx = 0.0f;
    float Fy = 0.0f;
    float Fz = 0.0f;
    for(int j = 0; j < nParticles; ++j) {
    	float dx = p[j].x - p[idx].x;
	float dy = p[j].y - p[idx].y;
      	float dz = p[j].z - p[idx].z;
      	float distSqr = dx*dx + dy*dy + dz*dz + SOFTENING;
      	float invDist = 1.0f / sqrtf(distSqr);
      	float invDist3 = invDist * invDist * invDist;
      	Fx += dx * invDist3; Fy += dy * invDist3; Fz += dz * invDist3;	
    }
    p[idx].vx += dt * Fx;
    p[idx].vy += dt * Fy;
    p[idx].vz += dt * Fz;  
}

__global__ void update_posn(Particle* p,  int nParticles, float dt) {
    int i = threadIdx.x + blockDim.x*blockIdx.x;
    if (i >= nParticles) return;
    p[i].x += p[i].vx*dt;
    p[i].y += p[i].vy*dt;
    p[i].z += p[i].vz*dt; 
}


int main(const int argc, const char** argv) {
  FILE *datafile    = NULL;      /* output file for particle positions */
  int   nParticles  = 300;      /* number of particles */
  int nIters = 200;
  if (argc > 1)
    nParticles      = atoi(argv[1]);

  if (argc > 2) 
    nIters         = atoi(argv[2]);
   int threads_per_block = 256; 
   float t ;
   hipEvent_t start, stop; 
   hipEventCreate(&start);
   hipEventCreate(&stop);
   int n_blocks = MIN(nParticles/threads_per_block + (nParticles % threads_per_block != 0) , MAX_BLOCKS_PER_DIM);

  const float dt    = 0.01f; /* time step   */

  float *buf_h        =  (float*)malloc(nParticles*sizeof(Particle));
  Particle  *p_h          = (Particle *) buf_h;
  
  ran_init(buf_h, 6*nParticles); /* Init pos and vel data */

  Particle *p_d;
  float *buf_d;
  hipMalloc((void**) &buf_d, nParticles * sizeof(Particle));
  p_d = (Particle *) buf_d;
  datafile          = fopen("particles_gpu.csv","w");

  float final_t = 0;

  for (int iter = 1; iter <= nIters; iter++) {
    printf("iteration:%d\n", iter);
    for (int i = 0;i < nParticles; ++i)
      fprintf(datafile, "%d, %f, %f, %f \n",iter, p_h[i].x, p_h[i].y, p_h[i].z);
    hipEventRecord(start, 0);
    hipMemcpy(buf_d, buf_h, nParticles * sizeof(Particle), hipMemcpyHostToDevice);
    calc_force<<<n_blocks, threads_per_block>>>(p_d, nParticles, dt);
    update_posn<<<n_blocks, threads_per_block>>>(p_d, nParticles, dt);
    assert((hipMemcpy(buf_h, buf_d, nParticles * sizeof(Particle), hipMemcpyDeviceToHost)) == hipSuccess);
    hipEventRecord(stop , 0);
    hipEventElapsedTime( &t, start, stop);
    final_t += t;
  }
  fclose(datafile);
  printf("avgTime: %f   totTime: %f \n", final_t/ (1000.0 * (nIters - 1)), final_t / 1000.0);
}

